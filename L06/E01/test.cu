// test.cu

#include <hip/hip_runtime.h>
#include <iostream>
__global__ void testKernel() {
    printf("Hello from kernel\n");
}
int main() {
    testKernel<<<1, 1>>>();
    hipDeviceSynchronize();
    return 0;
}
